#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include <chrono>

#include <stdio.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// note: functor inherits from unary_function
struct get_binaryIndex : public thrust::unary_function<unsigned int, unsigned int>
{
	__host__ __device__
		unsigned int operator()(unsigned int x) const
	{
		return x & 0xFFFF0000;
	}
};

// note: functor inherits from unary_function
struct get_setFlags : public thrust::unary_function<unsigned int, unsigned int>
{
	__host__ __device__
		unsigned int operator()(unsigned int x) const
	{
		return x & 0x0000FFFF;
	}
};

/*struct Compare_custom
{
	bool operator () (const Example& first, const Example& second)
	{
		if (first.a.size() > second.a.size())
			return true;
		else
			return false;
	}
};
*/
int main()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);
		std::cout << "Device: " << deviceProp.computeMode << " " << deviceProp.name << std::endl;
	}

	hipSetDevice(0);
	// generate 32M random numbers serially
	long long vectorSize = 600000000;
	thrust::host_vector<unsigned int> h_vec1(vectorSize);
	thrust::host_vector<unsigned int> h_vec2(vectorSize);
	thrust::host_vector<unsigned int> h_vec3(vectorSize);
	std::generate(h_vec1.begin(), h_vec1.end(), rand);
	std::generate(h_vec2.begin(), h_vec2.end(), rand);

	auto start = std::chrono::high_resolution_clock::now();
	// transfer data to the device
	thrust::device_vector<int> d_vec1 = h_vec1;
	thrust::device_vector<int> d_vec2 = h_vec2;

	// transfer data back to host
	thrust::copy(d_vec1.begin(), d_vec1.end(), h_vec1.begin());
	auto stop = std::chrono::high_resolution_clock::now();

	auto duration = std::chrono::duration_cast<std::chrono::seconds>(stop - start);

	// To get the value of duration use the count()
	// member function on the duration object
	std::cout << "Execution time in seconds: " << duration.count() << std::endl;

	return 0;
}